#include "hip/hip_runtime.h"
#ifndef __BITONIC_KERNEL_CU__
#define __BITONIC_KERNEL_CU__

#define CHECK_SYNC
#include "CudaErrorCheck.cu"
#include "bitonic_kernel.cuh"


template<typename T>
__global__ void bitonic_kernel(T *d_base, size_t num, int j, int k, 
                               int (*cmp)(const void *, const void *)) {
    int i, pair;
    T tmp, *i_ptr, *pair_ptr;

    i = threadIdx.x + blockIdx.x * blockDim.x;
    pair = i ^ j;
    if ( i >= num || i >= pair ) return;

    i_ptr = d_base + i;
    pair_ptr = d_base + pair;

    if ( ((i & k) == 0 && cmp(i_ptr, pair_ptr) > 0) ||
         ((i & k) != 0 && cmp(i_ptr, pair_ptr) < 0)) {
        tmp = *i_ptr;
        *i_ptr = *pair_ptr;
        *pair_ptr = tmp;
    }
}

template<typename T>
T maximum(T *base, size_t num) {
    T tmp, max = *base;

    while ( --num ) {
        tmp = *++base;
        if ( tmp > max ) max = tmp;
    }

    return max;
}

// Least power of 2 greater than or equal to x (from Hacker's Delight)
inline size_t clp2(size_t x) {
   x -= 1;
   x |= (x >> 1);
   x |= (x >> 2);
   x |= (x >> 4);
   x |= (x >> 8);
   x |= (x >> 16);

   return ++x;
}

template<typename T>
void bitonic_sort(T *base, size_t num,
                  int (*cmp)(const void *, const void *)) {
    T *d_base;
    hipDeviceProp_t prop;
    size_t boundary = clp2(num);
    std::cout << boundary << std::endl;
    T pad;

    cudaSafeCall( hipMalloc(&d_base, boundary * sizeof(T)) );
    cudaSafeCall (
        hipMemcpy(d_base, base, num * sizeof(T), hipMemcpyHostToDevice) );

    if ( boundary != num ) {
        pad = maximum<T>(base, num);

        std::cout << "Padding array with " << boundary - num << " more " 
                  << pad << " after first " << num << " elements" << std::endl;
        // Pad our array to the next highest power of two (elements)
        cudaSafeCall( hipMemset(d_base + num, pad, boundary - num) );
    }

    cudaSafeCall( hipGetDeviceProperties(&prop, 0) );
    dim3 threads(prop.maxThreadsPerBlock);
    dim3 blocks(max((int) boundary / prop.maxThreadsPerBlock, 1));

    for ( int k = 2; k <= boundary; k <<= 1 ) {
        std::cout << "k: " << k << std::endl;
        for ( int j = k >> 1; j > 0; j >>= 1 ) {
            std::cout << "j: " << j << std::endl;
            bitonic_kernel<<<blocks, threads>>>(d_base, boundary, j, k, cmp);
            cudaCheckError();
        }
    }

    cudaSafeCall( 
        hipMemcpy(base, d_base, num * sizeof(T), hipMemcpyDeviceToHost) );
    cudaSafeCall( hipFree(d_base) );
}

template void bitonic_sort(unsigned *, size_t, 
                           int (*cmp)(const void *, const void *));


#endif // __BITONIC_KERNEL_CU__