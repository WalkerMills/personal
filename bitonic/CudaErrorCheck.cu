/* Based on:
 *     https://gist.github.com/ashwin/2652488#file-cudaerrorcheck-cu
 */

#ifndef __CUDA_ERROR_CHECK_CU__
#define __CUDA_ERROR_CHECK_CU__

#include <iostream>
#include <cstdlib>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>


// Define this to turn on error checking
#define CUDA_ERROR_CHECK
// Define this to turn on synchronous error checking
// #define CHECK_SYNC

// Wrapper macros
#define cudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define curandSafeCall( err ) __cuSafeCall( err, HIPRAND_STATUS_SUCCESS, __FILE__, __LINE__ )
#define cublasSafeCall( err ) __cuSafeCall( err, HIPBLAS_STATUS_SUCCESS, __FILE__, __LINE__ )
#define cudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

// Safely make a CUDA API call
inline void __cudaSafeCall(hipError_t err, const char *file, const int line,
                           bool abort_=true) {
#ifdef CUDA_ERROR_CHECK
    if ( err != hipSuccess ) {
        std::cerr << "cudaSafeCall() failed at " << file << ":" << line
                  << " : " << hipGetErrorString(err) << std::endl;

        if ( abort_ ) exit(EXIT_FAILURE);
    }
#endif // CUDA_ERROR_CHECK

    return;
}

inline void __cuSafeCall(int err, int success, char *file, int line,
                         bool abort_=true) {
#ifdef CUDA_ERROR_CHECK
    if ( err != success ) {
        std::cerr << "cuSafeCall failed at " << file << ":" << line
                  << " with code " << err << std::endl;
                  
        if ( abort_ ) exit(EXIT_FAILURE);
    }
#endif
}

// Check if the last kernel call failed
inline void __cudaCheckError(const char *file, const int line,
                             bool abort_=true) {
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess ) {
        std::cerr << "cudaCheckError() failed at " << file << ":" << line
                  << " : " << hipGetErrorString(err) << std::endl;

        if ( abort_ ) exit(EXIT_FAILURE);
    }

#ifdef CHECK_SYNC
    // More careful checking. However, this will affect performance.
    err = cudaDeviceSynchronize();
    if ( err != cudaSuccess ) {
        std::cerr << "cudaCheckError() with sync failed at " << file << ":" 
                  << line << " : " << cudaGetErrorString(err) << std::endl;

        if ( abort_ ) exit(EXIT_FAILURE);
    }
#endif // CHECK_SYNC
    
#endif // CUDA_ERROR_CHECK

    return;
}

#endif // __CUDA_ERROR_CHECK_CU__